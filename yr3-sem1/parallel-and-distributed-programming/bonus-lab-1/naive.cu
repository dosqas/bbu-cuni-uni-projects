#include "hip/hip_runtime.h"
#include "naive.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <chrono>

using namespace std::chrono;

void Naive::Sequential(const int* poly1, const int* poly2, int degree1, int degree2, int* result) {
    std::cout << "\nRunning naive approach (sequential)..." << std::endl;
    auto start = high_resolution_clock::now();

    for (int i = 0; i < degree1; i++) {
        for (int j = 0; j < degree2; j++) {
            result[i + j] += poly1[i] * poly2[j];
        }
    }

    auto stop = high_resolution_clock::now();
    std::cout << "Time taken (naive sequential): "
        << duration_cast<milliseconds>(stop - start).count() << " ms\n";
}

// Function runs on the GPU, launched by the host (CPU)
__global__ void polyMultiplyKernelNaive(const int* poly1, const int* poly2, int* result, int degree1, int degree2) {
	// Calculate global thread indices
	// Each thread has acces to its unique indices within its block (threadIdx) and the block's indices within the grid (blockIdx)
	// To access its block, we multiply blockIdx by blockDim (number of threads per block)
	// and add threadIdx to get the index of the thread within the entire grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

	// If indices are within bounds, perform multiplication and accumulate result
	// To synchronize access to shared memory, we use atomicAdd to prevent
	// race conditions
    if (i < degree1 && j < degree2) {
        atomicAdd(&result[i + j], poly1[i] * poly2[j]);
    }
}

void Naive::CUDA(const int* h_poly1, const int* h_poly2, int degree1, int degree2, int* h_result) {
    int resultLength = degree1 + degree2 - 1;

    // Allocate device memory
	// The GPU has its own memory space, so we need to allocate memory on it
    int* d_poly1, * d_poly2, * d_result;
    hipMalloc(&d_poly1, degree1 * sizeof(int));
    hipMalloc(&d_poly2, degree2 * sizeof(int));
    hipMalloc(&d_result, resultLength * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_poly1, h_poly1, degree1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_poly2, h_poly2, degree2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, resultLength * sizeof(int));

    std::cout << "\nRunning naive approach (CUDA)..." << std::endl;
    auto start = high_resolution_clock::now();

    // Define CUDA grid and block dimensions
	// Thread - smallest units of execution
	// Block - group of threads that can cooperate with each other
	// Grid - group of blocks
	// 16 x 16 = 256 threads per block
	// polynomial_degree + 15 / 16 to round down to nearest integer (ensure enough blocks to cover all coefficients)
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((degree1 + 15) / 16, (degree2 + 15) / 16);

    // Launch kernel
	// <<< ... >>> = execution configuration syntax
	// We launch it with the specified number of blocks and threads per block
    polyMultiplyKernelNaive<<<numBlocks, threadsPerBlock>>>(d_poly1, d_poly2, d_result, degree1, degree2);

	// Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    auto stop = high_resolution_clock::now();
    std::cout << "Time taken (naive CUDA): "
        << duration_cast<milliseconds>(stop - start).count() << " ms\n";

    // Copy result back to host
    hipMemcpy(h_result, d_result, resultLength * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_poly1);
    hipFree(d_poly2);
    hipFree(d_result);
}
