#include "hip/hip_runtime.h"
#include "karatsuba.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <chrono>
#include <vector>
using namespace std;
using namespace std::chrono;

void Karatsuba::KaratsubaRecursiveNonParallel(const int* poly1, const int* poly2, int* result, int n) {
    if (n <= 32) {  
        for (int i = 0; i < n; i++)
            for (int j = 0; j < n; j++)
                result[i + j] += poly1[i] * poly2[j];
        return;
    }

    int k = n / 2;

    vector<int> z0(2 * k, 0), z1(2 * k, 0), z2(2 * k, 0);
    vector<int> sumA(k, 0), sumB(k, 0);

    // z0 = poly1Low * poly2Low
    KaratsubaRecursiveNonParallel(poly1, poly2, z0.data(), k);

    // z2 = poly1High * poly2High
    KaratsubaRecursiveNonParallel(poly1 + k, poly2 + k, z2.data(), k);

    // sumA = poly1Low + poly1High
    // sumB = poly2Low + poly2High
    for (int i = 0; i < k; i++) {
        sumA[i] = poly1[i] + poly1[i + k];
        sumB[i] = poly2[i] + poly2[i + k];
    }

    // z1 = (sumA * sumB)
    KaratsubaRecursiveNonParallel(sumA.data(), sumB.data(), z1.data(), k);

    // z1 = z1 - z0 - z2
    for (int i = 0; i < 2 * k; i++)
        z1[i] -= z0[i] + z2[i];

    // Combine results
    for (int i = 0; i < 2 * k; i++) result[i] += z0[i];
    for (int i = 0; i < 2 * k; i++) result[i + k] += z1[i];
    for (int i = 0; i < 2 * k; i++) result[i + 2 * k] += z2[i];
}

void Karatsuba::KaratsubaAlgorithm(const int* poly1, const int* poly2, int n, bool isSequential) {
    vector<int> result(2 * n, 0);

    if (isSequential) {
        std::cout << "\nRunning Karatsuba algorithm (sequential)..." << endl;
        auto start = high_resolution_clock::now();
        KaratsubaRecursiveNonParallel(poly1, poly2, result.data(), n);
        auto stop = high_resolution_clock::now();
        std::cout << "Time taken (Karatsuba sequential): "
            << duration_cast<milliseconds>(stop - start).count() << " ms\n";
    }
    else {
        std::cout << "\nRunning Karatsuba algorithm (CUDA)..." << endl;
		auto start = high_resolution_clock::now();
        KaratsubaCUDA(poly1, poly2, result.data(), n);
		auto stop = high_resolution_clock::now();
        std::cout << "Time taken (Karatsuba CUDA): "
			<< duration_cast<milliseconds>(stop - start).count() << " ms\n";
    }
}

__global__ void karatsubaSingleSplitKernel(
    const int* poly1, const int* poly2, int* result,
    int n, int k)
{
    // Access the thread's unique index within the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

	// Ensure we don't go out of bounds
	// (bounds = half the size of the polynomials)
    if (i < k) {
        // low/high halves
        int low1 = poly1[i];
        int high1 = poly1[i + k];
        int low2 = poly2[i];
        int high2 = poly2[i + k];

        atomicAdd(&result[i + i], low1 * low2); // z0
        atomicAdd(&result[i + i + k], low1 * high2 + high1 * low2); // z1
        atomicAdd(&result[i + i + 2 * k], high1 * high2); // z2
    }
}

void Karatsuba::KaratsubaCUDA(const int* h_poly1, const int* h_poly2, int* h_result, int n) {
    int* d_poly1, * d_poly2, * d_result;
    int resultSize = 2 * n - 1;

	// Allocate device memory
    hipMalloc(&d_poly1, n * sizeof(int));
    hipMalloc(&d_poly2, n * sizeof(int));
    hipMalloc(&d_result, resultSize * sizeof(int));

	// Initialize result array on device to zero
    hipMemset(d_result, 0, resultSize * sizeof(int));

    hipMemcpy(d_poly1, h_poly1, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_poly2, h_poly2, n * sizeof(int), hipMemcpyHostToDevice);

    int k = n / 2;
    int threads = 256;
    int blocks = (k + threads - 1) / threads;

    karatsubaSingleSplitKernel<<<blocks, threads>>>(d_A, d_B, d_C, n, k);
    hipDeviceSynchronize();

    hipMemcpy(h_result, d_result, resultSize * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_poly1);
    hipFree(d_poly2);
    hipFree(d_result);
}